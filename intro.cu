#include "hip/hip_runtime.h"
/*
 * 
 * Programa de Introducción a los conceptos de CUDA
 * 
 *
 * 
 * 
 */

#include <stdio.h>
#include <stdlib.h>

/* Declaración de métodos/


/* Utilidad para checar errores de CUDA */
void checkCUDAError(const char*);

/* Kernel para sumar dos vectores en un sólo bloque de hilos */
__global__ void vect_add(int *d_a, int *d_b, int *d_out)
{
    /* 
     * Part 2B: Implementación del kernel para realizar la suma de los vectores en el GPU
     * Revisado por Victor
     */
    int idx = threadIdx.x;
    int numA = d_a[idx];
    int numB = d_b[idx];
	d_out[idx] = numA + numB;
}

/* Versión de múltiples bloques de la suma de vectores */
__global__ void vect_add_multiblock(int *d_a, int *d_b, int *d_out)
{
    /* Part 2C: Implementación del kernel pero esta vez permitiendo múltiples bloques de hilos. */
    int idx = threadIdx.x + (blockIdx.x * blockDim.x);
    int numA = d_a[idx];
    int numB = d_b[idx];
	d_out[idx] = numA + numB;
}

/* Numero de elementos en el vector */
#define ARRAY_SIZE 256

/*
 * Número de bloques e hilos
 * Su producto siempre debe ser el tamaño del vector (arreglo).
 */
#define NUM_BLOCKS  1
#define THREADS_PER_BLOCK 256

/* Main routine */
int main(int argc, char *argv[])
{
    int *a, *b, *c; /* Arreglos del CPU */
    int *d_a, *d_b, *d_c;/* Arreglos del GPU */

    int i;
    size_t sz = ARRAY_SIZE * sizeof(int);

    /*
     * Reservar memoria en el cpu
     */
    a = (int *) malloc(sz);
    b = (int *) malloc(sz);
    c = (int *) malloc(sz);

    /*
     * Parte 1A:Reservar memoria en el GPU
     * Revisado por Victor
     */
    hipMalloc((void**) &d_a, sz);
    hipMalloc((void**) &d_b, sz);
    hipMalloc((void**) &d_c, sz);

    /* inicialización */
    for (i = 0; i < ARRAY_SIZE; i++) {
        a[i] = i;
        b[i] = ARRAY_SIZE - i;
        c[i] = 0;
    }

    /* 
     * Parte 1B: Copiar los vectores del CPU al GPU
     * Revisado por Victor
     */
    hipMemcpy(d_a, a, sz, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sz, hipMemcpyHostToDevice);

    /* run the kernel on the GPU */
    /*
     * Parte 2A: Configurar y llamar los kernels
     * Revisado por Victor
     */
    dim3 dimGrid(NUM_BLOCKS, ARRAY_SIZE);
    dim3 dimBlock(THREADS_PER_BLOCK);
    printf("\nSUMA en bloques\n");
    vect_add_multiblock<<<dimGrid, dimBlock>>>(d_a, d_b, d_c);
    // printf("\nSUMA simple\n");
    // vect_add<<<1, ARRAY_SIZE>>>(d_a, d_b, d_c);

    /* Esperar a que todos los threads acaben y checar por errores */
    hipDeviceSynchronize();
    checkCUDAError("kernel invocation");

    /* 
     * Part 1C: copiar el resultado de nuevo al CPU
     * Revisado por Victor
     */
    hipMemcpy(c, d_c, sz, hipMemcpyDeviceToHost);

    checkCUDAError("memcpy");

    /* print out the result */
    printf("Results: ");
    for (i = 0; i < ARRAY_SIZE; i++) {
      printf("%d, ", c[i]);
    }
    printf("\n\n");

    /* 
     * Parte 1D: Liberar los arreglos
     * Revisado por Victor
     */
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(a);
    free(b);
    free(c);

    return 0;
}


/* Utility function to check for and report CUDA errors */
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}
